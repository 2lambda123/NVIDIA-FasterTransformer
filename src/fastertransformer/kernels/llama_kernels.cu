#include "hip/hip_runtime.h"
#include "src/fastertransformer/kernels/llama_kernels.h"
#include "src/fastertransformer/utils/cuda_fp8_utils.h"

#include <algorithm>

#include <assert.h>
#include <hip/hip_fp16.h>
#include <stdio.h>

using namespace std;
namespace fastertransformer {

template<typename T>
__global__ void LLaMAstart_id_embedding_lookups_kernel(
    T* out, const T* embedding_table, const int* input_ids, const int num_tokens, const int64_t hidden_units)
{
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < num_tokens * hidden_units;
         index += blockDim.x * gridDim.x) {

        // embedding lookup from word ids [batch, length] (part of [batch, length]) and [vocab, hidden] to generate
        // embedding [batch, length, hidden]
        const int word_index = index / hidden_units;
        const int col_index  = index % hidden_units;
        const int input_id   = input_ids[word_index];

        out[index] = embedding_table[input_id * hidden_units + col_index];
    }
}

template<typename T>
void invokeLLaMAInputIdsEmbeddingLookup(T*           out,
                                        const T*     embedding_table,
                                        const int*   input_ids,
                                        const int    num_tokens,
                                        const int    hidden_units,
                                        hipStream_t stream)
{
    dim3 grid(min(num_tokens, 65536));
    dim3 block(min(hidden_units, 512));
    LLaMAstart_id_embedding_lookups_kernel<T>
        <<<grid, block, 0, stream>>>(out, embedding_table, input_ids, num_tokens, hidden_units);
}

template void invokeLLaMAInputIdsEmbeddingLookup(float*       out,
                                                 const float* embedding_table,
                                                 const int*   input_ids,
                                                 const int    num_tokens,
                                                 const int    hidden_units,
                                                 hipStream_t stream);
template void invokeLLaMAInputIdsEmbeddingLookup(half*        out,
                                                 const half*  embedding_table,
                                                 const int*   input_ids,
                                                 const int    num_tokens,
                                                 const int    hidden_units,
                                                 hipStream_t stream);

__global__ void LLaMAgetPaddingOffsetAndCuSeqLensKernel(
    int* padding_offset, int* cu_seqlens, const int* sequence_length, const int batch_size, const int seq_len)
{
    // do cumulated sum
    int total_seq_len = 0;
    int cum_offset    = 0;
    int index         = 0;
    for (int i = 0; i < batch_size; i++) {
        const int num_tokens = sequence_length[i];
        cu_seqlens[i]        = total_seq_len;
        for (int j = 0; j < num_tokens; j++) {
            padding_offset[index] = cum_offset;
            index++;
        }
        cum_offset += seq_len - num_tokens;
        total_seq_len += num_tokens;
    }
    cu_seqlens[batch_size] = total_seq_len;
}

void invokeLLaMAGetPaddingOffsetAndCuSeqLens(int*         padding_offset,
                                             int*         cu_seqlens,
                                             const int*   input_lengths,
                                             const int    batch_size,
                                             const int    seq_len,
                                             hipStream_t stream)
{
    LLaMAgetPaddingOffsetAndCuSeqLensKernel<<<1, 1, 0, stream>>>(
        padding_offset, cu_seqlens, input_lengths, batch_size, seq_len);
}

template<typename T>
__global__ void LLaMAbuildDecoderAttentionMaskKernel(T*         attention_mask,
                                                     const int* sequence_lengths,
                                                     const int* context_lengths,
                                                     const int  batch_size,
                                                     const int  seq_len,
                                                     const int  attn_len)
{
    // attention_mask:
    // [batch_size, 1, seq_len, attn_len]
    const int batch_idx         = blockIdx.x;
    const int mask_size_per_seq = seq_len * attn_len;
    attention_mask += batch_idx * mask_size_per_seq;
    const int context_length = context_lengths[batch_idx];
    const int length         = sequence_lengths[batch_idx];

    for (int i = threadIdx.x; i < mask_size_per_seq; i += blockDim.x) {
        int row_id = i / attn_len;
        int col_id = i % attn_len;
        if (row_id < length && col_id <= (row_id + context_length)) {
            attention_mask[i] = (T)(1.0f);
        }
        else {
            attention_mask[i] = (T)(0.0f);
        }
    }
}

template<typename T>
void invokeLLaMABuildDecoderAttentionMask(T*           attention_mask,
                                          const int*   sequence_length,
                                          const int*   context_lengths,
                                          const int    batch_size,
                                          const int    seq_len,
                                          const int    attn_len,
                                          hipStream_t stream)
{
    LLaMAbuildDecoderAttentionMaskKernel<T><<<batch_size, 256, 0, stream>>>(
        attention_mask, sequence_length, context_lengths, batch_size, seq_len, attn_len);
}

template void invokeLLaMABuildDecoderAttentionMask(float*       attention_mask,
                                                   const int*   sequence_length,
                                                   const int*   context_lengths,
                                                   const int    batch_size,
                                                   const int    seq_len,
                                                   const int    attn_len,
                                                   hipStream_t stream);

template void invokeLLaMABuildDecoderAttentionMask(half*        attention_mask,
                                                   const int*   sequence_length,
                                                   const int*   context_lengths,
                                                   const int    batch_size,
                                                   const int    seq_len,
                                                   const int    attn_len,
                                                   hipStream_t stream);

template<typename T>
__global__ void LLaMACopyKernel(T* dst, T* src, const int count)
{

    int           idx     = blockIdx.x * blockDim.x + threadIdx.x;
    constexpr int X_ELEMS = (sizeof(T) == 4) ? 4 : 8;
    if (idx * X_ELEMS >= count) {
        return;
    }

    auto v_dst = reinterpret_cast<uint4*>(dst);
    auto v_src = reinterpret_cast<uint4*>(src);
    v_dst[idx] = v_src[idx];
}

template<typename T>
void invokeLLaMACopyKernel(T* dst, T* src, const int count, hipStream_t stream)
{
    constexpr int block_sz = 128;
    constexpr int x        = (sizeof(T) == 4) ? 4 : 8;
    assert(count % x == 0);
    int grid_sz = (count / x + block_sz - 1) / block_sz;
    LLaMACopyKernel<<<grid_sz, block_sz, 0, stream>>>(dst, src, count);
}

template void invokeLLaMACopyKernel(float* dst, float* src, const int count, hipStream_t stream);
template void invokeLLaMACopyKernel(half* dst, half* src, const int count, hipStream_t stream);

}  // namespace fastertransformer
