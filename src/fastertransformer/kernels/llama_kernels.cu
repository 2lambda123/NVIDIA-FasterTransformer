#include "hip/hip_runtime.h"
#include "src/fastertransformer/kernels/llama_kernels.h"
#include "src/fastertransformer/utils/cuda_fp8_utils.h"

#include <assert.h>
#include <hip/hip_fp16.h>
#include <stdio.h>

namespace fastertransformer {

__global__ void LLaMAgetPaddingOffsetAndCuSeqLensKernel(
    int* padding_offset, int* cu_seqlens, const int* sequence_length, const int batch_size, const int seq_len)
{
    // do cumulated sum
    int total_seq_len = 0;
    int cum_offset    = 0;
    int index         = 0;
    for (int i = 0; i < batch_size; i++) {
        const int num_tokens = sequence_length[i];
        cu_seqlens[i]        = total_seq_len;
        for (int j = 0; j < num_tokens; j++) {
            padding_offset[index] = cum_offset;
            index++;
        }
        cum_offset += seq_len - num_tokens;
        total_seq_len += num_tokens;
    }
    cu_seqlens[batch_size] = total_seq_len;
}

void invokeLLaMAGetPaddingOffsetAndCuSeqLens(int*         padding_offset,
                                             int*         cu_seqlens,
                                             const int*   input_lengths,
                                             const int    batch_size,
                                             const int    seq_len,
                                             hipStream_t stream)
{
    LLaMAgetPaddingOffsetAndCuSeqLensKernel<<<1, 1, 0, stream>>>(
        padding_offset, cu_seqlens, input_lengths, batch_size, seq_len);
}

template<typename T>
__global__ void LLaMAbuildDecoderAttentionMaskKernel(T*         attention_mask,
                                                     const int* sequence_lengths,
                                                     const int* context_lengths,
                                                     const int  batch_size,
                                                     const int  seq_len,
                                                     const int  max_length)
{
    // attention_mask:
    // [batch_size, 1, seq_len, max_length]
    const int batch_idx         = blockIdx.x;
    const int mask_size_per_seq = seq_len * max_length;
    attention_mask += batch_idx * mask_size_per_seq;
    const int context_length = context_lengths[batch_idx];
    const int length         = sequence_lengths[batch_idx];
    const int offset         = max_length - length;

    for (int i = threadIdx.x; i < mask_size_per_seq; i += blockDim.x) {
        int row_id = i / max_length;
        int col_id = i % max_length;
        if (row_id < length && col_id <= (row_id + context_length)) {
            attention_mask[i] = (T)(1.0f);
        }
        else {
            attention_mask[i] = (T)(0.0f);
        }
    }
}

template<typename T>
void invokeLLaMABuildDecoderAttentionMask(T*           attention_mask,
                                          const int*   sequence_length,
                                          const int*   context_lengths,
                                          const int    batch_size,
                                          const int    seq_len,
                                          const int    max_length,
                                          hipStream_t stream)
{
    LLaMAbuildDecoderAttentionMaskKernel<T><<<batch_size, 256, 0, stream>>>(
        attention_mask, sequence_length, context_lengths, batch_size, seq_len, max_length);
}

template void invokeLLaMABuildDecoderAttentionMask(float*       attention_mask,
                                                   const int*   sequence_length,
                                                   const int*   context_lengths,
                                                   const int    batch_size,
                                                   const int    seq_len,
                                                   const int    max_length,
                                                   hipStream_t stream);

template void invokeLLaMABuildDecoderAttentionMask(half*        attention_mask,
                                                   const int*   sequence_length,
                                                   const int*   context_lengths,
                                                   const int    batch_size,
                                                   const int    seq_len,
                                                   const int    max_length,
                                                   hipStream_t stream);

template<typename T>
__global__ void LLaMACopyKernel(T* dst, T* src, const int count)
{

    int           idx     = blockIdx.x * blockDim.x + threadIdx.x;
    constexpr int X_ELEMS = (sizeof(T) == 4) ? 4 : 8;
    if (idx * X_ELEMS >= count) {
        return;
    }

    auto v_dst = reinterpret_cast<uint4*>(dst);
    auto v_src = reinterpret_cast<uint4*>(src);
    v_dst[idx] = v_src[idx];
}

template<typename T>
void invokeLLaMACopyKernel(T* dst, T* src, const int count, hipStream_t stream)
{
    constexpr int block_sz = 128;
    constexpr int x        = (sizeof(T) == 4) ? 4 : 8;
    assert(count % x == 0);
    int grid_sz = (count / x + block_sz - 1) / block_sz;
    LLaMACopyKernel<<<grid_sz, block_sz, 0, stream>>>(dst, src, count);
}

template void invokeLLaMACopyKernel(float* dst, float* src, const int count, hipStream_t stream);
template void invokeLLaMACopyKernel(half* dst, half* src, const int count, hipStream_t stream);

}  // namespace fastertransformer
